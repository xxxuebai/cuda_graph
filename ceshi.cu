#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>

const int N = 1024 * 1024;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256; 
__global__ void kernelA(int *a,int *b,int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
	
	}
}

__global__ void kernelB(int *a)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
	//	r[idx] = (c[idx] +a[idx])/2;
	a[idx] = a[idx] +1;
	}
}

int main()
{
	hipStream_t stream;
	hipStreamCreate(&stream);
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	//hipMalloc((void**)&dev_r,N*sizeof(int));

	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice,stream);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice,stream);
	for(i=0;i<100;i++)
	{
		kernelA << <N/GPUBLOCKNUM , GPUTHREADNUM,0,stream >> >(dev_a,dev_b,dev_c);
		kernelB << <N/GPUBLOCKNUM,GPUTHREADNUM,0,stream >> >(dev_a);
	}
	hipMemcpyAsync(host_c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost,stream);
	hipStreamSynchronize(stream);
	for(i=0;i<10;i++)
	{
		printf("%d ",host_c[i]);
	}

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipStreamDestroy(stream);
}
