/*
 源链接：
 https://blog.csdn.net/smartcat2010/article/details/105167981
 https://developer.nvidia.com/blog/cuda-graphs/
*/

//初始
#define NSTEP 1000
#define NKERNEL 20
 
// start CPU wallclock timer
for(int istep=0; istep<NSTEP; istep++){
  for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
    shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    hipStreamSynchronize(stream);
  }
}
//end CPU wallclock time

/*
总共平均耗时9.6μs；kernel执行耗时2.9us；
缺点：启动kernel-->执行kernel-->等待执行完；

device和host是异步的，当CPU调用device函数后就返回了;
hipMemcpy函数是个同步函数。
hipError_t hipMemcpyAsync(void* dst, const void* src, size_t count,hipMemcpyKind kind, hipStream_t stream = 0);
值得注意的就是最后一个参数，stream表示流，一般情况设置为默认流，这个函数和主机是异步的，执行后控制权立刻归还主机，
*/

//改进
// start wallclock timer
for(int istep=0; istep<NSTEP; istep++){
  for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
    shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
  }
  hipStreamSynchronize(stream);
}
//end wallclock timer

/*
 总共平均耗时3.8μs；kernel执行耗时2.9us；
 优点：启动下一个kernel和执行上一个kernel，能够并行起来；
 缺点：每个kernel还得启动一次；
*/


//Graph优化版本：
bool graphCreated=false;
hipGraph_t graph;
hipGraphExec_t instance;
for(int istep=0; istep<NSTEP; istep++){
  if(!graphCreated){
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
      shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    }
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    graphCreated=true;
  }
  hipGraphLaunch(instance, stream);
  hipStreamSynchronize(stream);
}
//总共平均耗时3.4μs；kernel执行耗时2.9us；
//优点：整个graph启动一次；头一次构建graph慢，但是后面的迭代就可以复用该graph了；

