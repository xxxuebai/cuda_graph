#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>

const int N = 1024 * 1024;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256; 
__global__ void kernelA(int *a,int *b,int *c)//核函数A 两个数组平均数
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
	}
}

__global__ void kernelB(int *c)//核函数B，A的结果+1
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = c[idx] +1;
	}
}
void cudaGraphSample(int *dev_a,int *dev_b,int *dev_c)//图任务建立
{
	hipStream_t stream0,streamForGraph;//定义两个流
	hipGraph_t graph;//定义一个图
	hipStreamCreate(&stream0);//创建流
	hipStreamCreate(&streamForGraph);//创建流
	hipStreamBeginCapture(stream0);//开始流捕获模式
	kernelA << <N/GPUBLOCKNUM , GPUTHREADNUM,0,stream0 >> >(dev_a,dev_b,dev_c);//执行核函数A
        kernelB << <N/GPUBLOCKNUM,GPUTHREADNUM,0,stream0 >> >(dev_c);//执行核函数B
	hipStreamEndCapture(stream0,&graph);//结束流捕获模式
	hipGraphExec_t graphExec;//创建一个实例
	hipGraphInstantiate(&graphExec,graph,NULL,NULL,0);//实例化
	hipGraphLaunch(graphExec,streamForGraph);//启动图
	hipStreamSynchronize(streamForGraph);//同步
	hipGraphExecDestroy(graphExec);//销毁
	hipGraphDestroy(graph);
	hipStreamDestroy(streamForGraph);
	hipStreamDestroy(stream0);
}

int main()
{
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));

	hipMalloc((void**)&dev_c,N*sizeof(int));
	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	cudaGraphSample(dev_a,dev_b,dev_c);
	hipMemcpyAsync(host_c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
//	for(i=0;i<20;i++)
//	{
//		for(int j=0;j<N;j++)
//	{
		printf("%d ",host_c[0]);
//		}
//		printf("\n");
//		}
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
