#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>
#include<vector>
const int N = 1024 * 1024;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256; 
__global__ void kernelA(int *a,int *b,int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
		
	}
}

__global__ void kernelB(int *a)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		//r[idx] = (c[idx] +a[idx]) /2;
	a[idx] = a[idx] + 1;
	}
}
void cudaGraphSample(int *dev_a,int *dev_b,int *dev_c)
{
	int i;
	hipStream_t streamForGraph;
	hipGraph_t graph;
	std::vector<hipGraphNode_t> nodeDependencies;
	hipGraphNode_t kernelNode;
	hipStreamCreate(&streamForGraph);

	hipKernelNodeParams kernelNodeParams = {0};
	
	hipGraphCreate(&graph,0);
	
	void *kernelArgs[3] = {(void *)&dev_a,(void *)&dev_b,(void *)dev_c};
	kernelNodeParams.func = (void *)kernelA;
	kernelNodeParams.gridDim = dim3(1,1,1);
	kernelNodeParams.blockDim = dim3(N/GPUBLOCKNUM,1,1);
	kernelNodeParams.sharedMemBytes = 0;
	kernelNodeParams.kernelParams =(void **) kernelArgs;

	kernelNodeParams.extra = NULL;

	hipGraphAddKernelNode(&kernelNode,graph,nodeDependencies.data(),nodeDependencies.size(),&kernelNodeParams);
	
	//nodeDependencies.clear();
	nodeDependencies.push_back(kernelNode);

	kernelNodeParams.func = (void *) kernelB;
	kernelNodeParams.gridDim = dim3(1,1,1);
        kernelNodeParams.blockDim = dim3(N/GPUBLOCKNUM,1,1);
        kernelNodeParams.sharedMemBytes = 0;
	void *kernelArgs2[1] = {(void *)&dev_a};
        kernelNodeParams.kernelParams =(void **)kernelArgs2;
	 kernelNodeParams.extra = NULL;
	hipGraphAddKernelNode(&kernelNode,graph,nodeDependencies.data(),nodeDependencies.size(),&kernelNodeParams);
	 nodeDependencies.clear();
        nodeDependencies.push_back(kernelNode);

	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec,graph,NULL,NULL,0);
	for(i=0;i<10;i++)
	{
		hipGraphLaunch(graphExec,streamForGraph);
	}
	hipStreamSynchronize(streamForGraph);
	hipGraphExecDestroy(graphExec);
	hipGraphDestroy(graph);
	hipStreamDestroy(streamForGraph);
}

int main()
{
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	//cudaMalloc((void**)&dev_r,N*sizeof(int));
	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	
		cudaGraphSample(dev_a,dev_b,dev_c);
	
	hipMemcpyAsync(host_c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(i=0;i<10;i++)
	{
		printf("%d ",host_c[i]);
	}
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
