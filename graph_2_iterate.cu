#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>

const int N = 1024 * 1024;
const int THREAD = 256;
const int BLOCK = 256; 
__global__ void kernelA(int *a,int *b,int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
                a[idx] = a[idx] + 1;
                b[idx] = b[idx] + 1;
	}
}

void cudaGraphSample(int *dev_a,int *dev_b,int *dev_c)
{
	hipStream_t stream0;
	hipGraph_t graph;
	hipStreamCreate(&stream0);
	hipStreamBeginCapture(stream0);
	kernelA << <N/BLOCK,THREAD,0,stream0 >> >(dev_a,dev_b,dev_c);
        kernelA << <N/BLOCK,THREAD,0,stream0 >> >(dev_a,dev_b,dev_c);
	hipStreamEndCapture(stream0,&graph);
	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec,graph,NULL,NULL,0);
	for(int i=0;i<100;i++)
	{
		hipGraphLaunch(graphExec,stream0);
	}
	hipStreamSynchronize(stream0);
	hipGraphExecDestroy(graphExec);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream0);
}

int main()
{
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	cudaGraphSample(dev_a,dev_b,dev_c);
	hipMemcpyAsync(host_c,dev_r,N * sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(i=0;i<10;i++)
	{
		printf("%d ",host_c[i]);
	}
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
