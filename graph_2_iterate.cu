#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>

const int N = 1024 * 1024;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256; 
__global__ void kernelA(int *a,int *b,int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
	}
}

__global__ void kernelB(int *c,int *r,int *a)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		r[idx] = (c[idx] +a[idx]) /2;
	}
}
void cudaGraphSample(int *dev_a,int *dev_b,int *dev_c,int *dev_r)
{
	hipStream_t stream0;
	hipGraph_t graph;
	hipStreamCreate(&stream0);
//	hipStreamCreate(&streamForGraph);
	hipStreamBeginCapture(stream0);
	kernelA << <N/GPUBLOCKNUM , GPUTHREADNUM,0,stream0 >> >(dev_a,dev_b,dev_c);
        kernelB << <N/GPUBLOCKNUM,GPUTHREADNUM,0,stream0 >> >(dev_c,dev_r,dev_a);
	hipStreamEndCapture(stream0,&graph);
	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec,graph,NULL,NULL,0);
	for(int i=0;i<100;i++)
	{
		hipGraphLaunch(graphExec,stream0);
	
	}
	hipStreamSynchronize(stream0);
	hipGraphExecDestroy(graphExec);
	hipGraphDestroy(graph);
	//hipStreamDestroy(streamForGraph);
	hipStreamDestroy(stream0);
}

int main()
{
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c,*dev_r;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	hipMalloc((void**)&dev_r,N*sizeof(int));
	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	
		cudaGraphSample(dev_a,dev_b,dev_c,dev_r);
	
	hipMemcpyAsync(host_c,dev_r,N * sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(i=0;i<10;i++)
	{
		printf("%d ",host_c[i]);
	}
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_r);
	hipFree(dev_c);
}
