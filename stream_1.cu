#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<math.h>

const int N = 1024 * 1024;
const int GPUTHREADNUM = 256;
const int GPUBLOCKNUM = 256; 
__global__ void kernelA(int *a,int *b,int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx])/2;
	}
}

__global__ void kernelB(int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = c[idx] +1;
	}
}

int main()
{
	hipStream_t stream;
	hipStreamCreate(&stream);
	int *host_a,*host_b,*host_c;
	int *dev_a,*dev_b,*dev_c;
	int i;
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));
	hipHostAlloc((void **)&host_a,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_b,N * sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void **)&host_c,N * sizeof(int),hipHostMallocDefault);
	for(i=0;i<N;i++)
  	{
		host_a[i] = N - i;
		host_b[i] = i;
	}
	hipMemcpyAsync(dev_a,host_a,N * sizeof(int),hipMemcpyHostToDevice,stream);
 	hipMemcpyAsync(dev_b,host_b,N * sizeof(int),hipMemcpyHostToDevice,stream);
	kernelA << <N/GPUBLOCKNUM , GPUTHREADNUM,0,stream >> >(dev_a,dev_b,dev_c);
	kernelB << <N/GPUBLOCKNUM,GPUTHREADNUM,0,stream >> >(dev_c);
	hipMemcpyAsync(host_c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost,stream);
	hipStreamSynchronize(stream);
//	for(i=0;i<20;i++)
//	{
//		for(int j=0;j<N;j++)
//	{
		printf("%d ",host_c[0]);
//		}
//		printf("\n");
//		}
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipStreamDestroy(stream);
}
